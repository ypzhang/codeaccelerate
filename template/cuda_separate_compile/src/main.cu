
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void child_thread()
{
 printf("inside child thread");
}

__global__ void main_kernel()
{
 printf("inside main thread");
 child_thread<<<1, 1>>>();
}

int main() 
{
 main_kernel<<<1, 1>>>();
}